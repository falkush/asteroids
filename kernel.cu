#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>

static uint8_t* buffer = 0;
static double* vecl = 0;
static int** gsyms = 0;
static int** ggstruct = 0;
static int** gperm = 0;
static int** gsign = 0;
static int** topo = 0;

static int* astnum = 0;
static int* lidx = 0;
static int* astnumplus = 0;
static double* astrad = 0;
static int* asttopo = 0;
static double* astpos0 = 0;
static double* astpos1 = 0;
static double* astpos2 = 0;
static int* astplusidx = 0;
static int* astsym = 0;
static double** astmat = 0;
static double** astmatv = 0;

static double* astv0 = 0;
static double* astv1 = 0;
static double* astv2 = 0;

static int* cnumplus = 0;
static double* ccpos0 = 0;
static double* ccpos1 = 0;
static double* ccpos2 = 0;
static int* ccsym = 0;

static int* lnumplus = 0;
static double* lpos0 = 0;
static double* lpos1 = 0;
static double* lpos2 = 0;
static double* lv0 = 0;
static double* lv1 = 0;
static double* lv2 = 0;

static double* seedist = 0;
static double* dist = 0;

static bool* lout = 0;
static int* loutf = 0;

static int* gginv = 0;

static bool* rip = 0;

static double* haz = 0;


__device__ void matmult(double* m1, double* m2)
{
	double m3[9]{};
	m3[0] = m1[0] * m2[0] + m1[1] * m2[3] + m1[2] * m2[6];
	m3[1] = m1[0] * m2[1] + m1[1] * m2[4] + m1[2] * m2[7];
	m3[2] = m1[0] * m2[2] + m1[1] * m2[5] + m1[2] * m2[8];
	m3[3] = m1[3] * m2[0] + m1[4] * m2[3] + m1[5] * m2[6];
	m3[4] = m1[3] * m2[1] + m1[4] * m2[4] + m1[5] * m2[7];
	m3[5] = m1[3] * m2[2] + m1[4] * m2[5] + m1[5] * m2[8];
	m3[6] = m1[6] * m2[0] + m1[7] * m2[3] + m1[8] * m2[6];
	m3[7] = m1[6] * m2[1] + m1[7] * m2[4] + m1[8] * m2[7];
	m3[8] = m1[6] * m2[2] + m1[7] * m2[5] + m1[8] * m2[8];

	for(int i=0;i<9;i++) m2[i] = m3[i];
}

__device__ int rnbw(int nbframe)
{
	int r = 0, g = 0, b = 0;
	double tmp;
	double x;

	x = fmod(nbframe * 0.006, 1.0);
	tmp = fmod(x, 1.0 / 6.0);

	if (x < 1.0 / 6.0)
	{
		r = 255;
		g = 1530 * tmp;
	}
	else if (x < 1.0 / 3.0)
	{
		g = 255;
		r = 255 - 1530 * tmp;
	}
	else if (x < 0.5)
	{
		g = 255;
		b = 1530 * tmp;
	}
	else if (x < 2.0 / 3.0)
	{
		b = 255;
		g = 255 - 1530 * tmp;
	}
	else if (x < 5.0 / 6.0)
	{
		b = 255;
		r = 1530 * tmp;
	}
	else
	{
		r = 255;
		b = 255 - 1530 * tmp;
	}

	return r + 256 * g + 256 * 256 * b;
}

__device__ int rnbw2(double x)
{
	int r = 0, g = 0, b = 0;
	double tmp;

	tmp = fmod(x, 1.0 / 6.0);

	if (x < 1.0 / 6.0)
	{
		r = 255;
		g = 1530 * tmp;
	}
	else if (x < 1.0 / 3.0)
	{
		g = 255;
		r = 255 - 1530 * tmp;
	}
	else if (x < 0.5)
	{
		g = 255;
		b = 1530 * tmp;
	}
	else if (x < 2.0 / 3.0)
	{
		b = 255;
		g = 255 - 1530 * tmp;
	}
	else if (x < 5.0 / 6.0)
	{
		b = 255;
		r = 1530 * tmp;
	}
	else
	{
		r = 255;
		b = 255 - 1530 * tmp;
	}

	return r + 256 * g + 256 * 256 * b;
}

__global__ void setast(uint8_t* buffer, double* astrad, double* astpos0, double* astpos1, double* astpos2, double* astv0, double* astv1, double* astv2, int* astnum, double* seedist, double* dist, double** astmat, double** astmatv)
{
	int i,l;
	int rand = 1;
	double maxang = 0.005;
	double tmp;
	double tmpmat[9]{};


	seedist[0] = 5;
	dist[0] = 2;
	astnum[0] = 1;

	astrad[0] = 0.2;
	astpos0[0] = 0;
	astpos1[0] = 0;
	astpos2[0] = 0;
	astv0[0] =0.001;
	astv1[0] = 0;
	astv2[0] =0;

	for (i = 0; i < 32; i++)
	{
		astmat[i][0] = 1;
		astmat[i][1] = 0;
		astmat[i][2] = 0;
		astmat[i][3] = 0;
		astmat[i][4] = 1;
		astmat[i][5] = 0;
		astmat[i][6] = 0;
		astmat[i][7] = 0;
		astmat[i][8] = 1;

		astmatv[i][0] = 1;
		astmatv[i][1] = 0;
		astmatv[i][2] = 0;
		astmatv[i][3] = 0;
		astmatv[i][4] = 1;
		astmatv[i][5] = 0;
		astmatv[i][6] = 0;
		astmatv[i][7] = 0;
		astmatv[i][8] = 1;

		for (l = 0; l < 10; l++) rand = (60493 * rand + 11) % 479001599;

		tmp = (2.0 * maxang) * (rand / 479001598.0) - maxang;

		tmpmat[0] = 1;
		tmpmat[1] = 0;
		tmpmat[2] = 0;
		tmpmat[3] = 0;
		tmpmat[4] = cos(tmp);
		tmpmat[5] = -sin(tmp);
		tmpmat[6] = 0;
		tmpmat[7] = sin(tmp);
		tmpmat[8] = cos(tmp);

		matmult(tmpmat, astmatv[i]);

		for (l = 0; l < 10; l++) rand = (60493 * rand + 11) % 479001599;

		tmp = (2.0 * maxang) * (rand / 479001598.0) - maxang;

		tmpmat[0] = cos(tmp);
		tmpmat[1] = 0;
		tmpmat[2] = sin(tmp);
		tmpmat[3] = 0;
		tmpmat[4] = 1;
		tmpmat[5] = 0;
		tmpmat[6] = -sin(tmp);
		tmpmat[7] = 0;
		tmpmat[8] = cos(tmp);

		matmult(tmpmat, astmatv[i]);

		for (l = 0; l < 10; l++) rand = (60493 * rand + 11) % 479001599;

		tmp = (2.0 * maxang) * (rand / 479001598.0) - maxang;

		tmpmat[0] = cos(tmp);
		tmpmat[1] = -sin(tmp);
		tmpmat[2] = 0;
		tmpmat[3] = sin(tmp);
		tmpmat[4] = cos(tmp);
		tmpmat[5] = 0;
		tmpmat[6] = 0;
		tmpmat[7] = 0;
		tmpmat[8] = 1;

		matmult(tmpmat, astmatv[i]);

	}
	
	for (i = 0; i < 4 * 1920 * 1080; i++) buffer[i] = 255;
	
}

__global__ void rotast(double** astmat, double** astmatv)
{
	int tmp = blockIdx.x * blockDim.x + threadIdx.x;

	matmult(astmatv[tmp], astmat[tmp]);
}

__global__ void modfovp(double* dist, bool fovp, bool fovm, bool resetflag)
{
	if (resetflag) dist[0] = 2.0;
	else
	{
		if (fovp) dist[0] += 0.05;
		else
		{
			if (dist[0] > 0.05) dist[0] -= 0.05;
		}
	}
}

__global__ void modfov(double* dist, double* vecl)
{
	int tmp = blockIdx.x * blockDim.x + threadIdx.x;
	int tmpx = tmp % 1920;
	int tmpy = (tmp - tmpx) / 1920;
		
		double sqsz = 0.01 / 6;

		double vec0, vec1, vec2;
		double addy0, addy1, addy2;
		double addz0, addz1, addz2;
		double vecn0, vecn1, vecn2;
		double x00 = 1, x01 = 0, x02 = 0;
		double x10 = 0, x11 = 1, x12 = 0;
		double x20 = 0, x21 = 0, x22 = 1;
		double multy = (1 - 1920) * sqsz / 2;
		double multz = (1080 - 1) * sqsz / 2;

		vec0 = dist[0] * x00 + multy * x10 + multz * x20;
		vec1 = dist[0] * x01 + multy * x11 + multz * x21;
		vec2 = dist[0] * x02 + multy * x12 + multz * x22;

		addy0 = sqsz * x10;
		addy1 = sqsz * x11;
		addy2 = sqsz * x12;

		addz0 = -sqsz * x20;
		addz1 = -sqsz * x21;
		addz2 = -sqsz * x22;


			vecn0 = vec0 + tmpx * addy0 + tmpy * addz0;
			vecn1 = vec1 + tmpx * addy1 + tmpy * addz1;
			vecn2 = vec2 + tmpx * addy2 + tmpy * addz2;

			vecl[tmp] = 1.0 / sqrt(vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2);
		
}

__global__ void modast(double* astrad, double* astpos0, double* astpos1, double* astpos2, double* astv0, double* astv1, double* astv2, int** topo, int** gperm, int** gsign, int currenttopo, int* astnum, int* astnumplus, double* lpos0, double* lpos1, double* lpos2, double* lv0, double* lv1, double* lv2, int* lnumplus, double pos0, double pos1, double pos2, double* ccpos0, double* ccpos1, double* ccpos2, int* cnumplus, bool fireflag, double v0, double v1, double v2, bool* lout, int* loutf, int nbframe, int* gginv, int* astplusidx, bool* rip, double* haz, int* lidx,bool resetflag, double* seedist, bool sdp, bool sdm, double cv0, double cv1, double cv2, bool w0, bool w1, bool w2, bool w3, bool w4, bool w5, int* ccsym, int* astsym, double** astmat, double** astmatv, int* asttopo)
{
	double lsize = 0.01;
	
	int l;
	int j,ii;
	int newsym;
	int ppos0, ppos1, ppos2;
	double postmp[3]{};
	double nastv[3]{};
	double nnastv[3]{};
	double disto;
	double theta, phi;
	double coord0, coord1, coord2;
	double dpos0, dpos1, dpos2;
	double tmp0, tmp1, tmp2;
	double csize = 0.05;
	int i, k;
	int astnumplustmp = 0;
	int cnumplustmp = 1;
	int lnumplustmp = 0;
	int tmpidx;
	double hazmax = 0;
	double tmpd;
	int tmpproj, projpp1, projpp2;
	double tmpmat;

	bool wall[6]{};

	wall[0] = w0;
	wall[1] = w1;
	wall[2] = w2;
	wall[3] = w3;
	wall[4] = w4;
	wall[5] = w5;

	if (sdp) seedist[0] += 0.1;

	if (sdm)
	{
		if (seedist[0] > 0.1) seedist[0] -= 0.1;
	}

	
	if (resetflag)
	{
		rip[0] = false;

		astnum[0] = 1;
		seedist[0] = 5;
		astrad[0] = 0.2;
		astpos0[0] = 0;
		astpos1[0] = 0;
		astpos2[0] = 0;
		astv0[0] = 0.001;
		astv1[0] = 0;
		astv2[0] = 0;

		lout[0] = false;
		lout[1] = false;
		lout[2] = false;
	}
	
	if (fireflag)
	{
		if (!lout[0])
		{
			lout[0] = true;
			loutf[0] = nbframe;

			lv0[0] = cv0+v0 / 50;
			lv1[0] =cv1+ v1 / 50;
			lv2[0] = cv2+ v2 / 50;

			lpos0[0] = pos0 - 0.5  -lv0[0];
			lpos1[0] = pos1 - 0.5 -lv1[0];
			lpos2[0] = pos2 - 0.5 -lv2[0];
		}
		else if (!lout[1])
		{
			lout[1] = true;
			loutf[1] = nbframe;

			lv0[1] = cv0+ v0 / 50;
			lv1[1] = cv1+ v1 / 50;
			lv2[1] = cv2+ v2 / 50;

			lpos0[1] = pos0 - 0.5  - lv0[1];
			lpos1[1] = pos1 - 0.5 - lv1[1];
			lpos2[1] = pos2 - 0.5  - lv2[1];
		}
		else if (!lout[2])
		{
			lout[2] = true;
			loutf[2] = nbframe;

			lv0[2] = cv0+ v0 / 50;
			lv1[2] = cv1+ v1 / 50;
			lv2[2] = cv2+ v2 / 50;

			lpos0[2] = pos0 - 0.5  - lv0[2];
			lpos1[2] = pos1 - 0.5  - lv1[2];
			lpos2[2] = pos2 - 0.5  - lv2[2];
		}
	}

	for (i = 0; i < 3; i++) if (lout[i] && nbframe - loutf[i] > 200) lout[i] = false;
	

	for (l = 0; l < astnum[0]; l++)
	{

		astpos0[l] += astv0[l];
		astpos1[l] += astv1[l];
		astpos2[l] += astv2[l];

		if (wall[0])
		{
			tmpd = astpos2[l] + astrad[l] - 0.5;
			if (tmpd > 0)
			{
				astpos2[l] -= tmpd;
				astv2[l] *= -1;
			}
		}
		if (wall[1])
		{
			tmpd = astpos0[l] + astrad[l] - 0.5;
			if (tmpd > 0)
			{
				astpos0[l] -= tmpd;
				astv0[l] *= -1;
			}
		}
		if (wall[2])
		{
			tmpd = astpos1[l] + astrad[l] - 0.5;
			if (tmpd > 0)
			{
				astpos1[l] -= tmpd;
				astv1[l] *= -1;
			}
		}
		if (wall[5])
		{
			tmpd = astpos2[l] - astrad[l] + 0.5;
			if (tmpd < 0)
			{
				astpos2[l] -= tmpd;
				astv2[l] *= -1;
			}
		}
		if (wall[3])
		{
			tmpd = astpos0[l] - astrad[l] + 0.5;
			if (tmpd < 0)
			{
				astpos0[l] -= tmpd;
				astv0[l] *= -1;
			}
		}
		if (wall[4])
		{
			tmpd = astpos1[l] - astrad[l] + 0.5;
			if (tmpd < 0)
			{
				astpos1[l] -= tmpd;
				astv1[l] *= -1;
			}
		}

		dpos0 = astpos0[l] + 0.5;
		dpos1 = astpos1[l] + 0.5;
		dpos2 = astpos2[l] + 0.5;

		ppos0 = dpos0;
		ppos1 = dpos1;
		ppos2 = dpos2;

		if (dpos0 < 0) ppos0--;
		if (dpos1 < 0) ppos1--;
		if (dpos2 < 0) ppos2--;

		if (ppos0 != 0 || ppos1 != 0 || ppos2 != 0)
		{
			ppos0 %= 12;
			ppos1 %= 12;
			ppos2 %= 12;
			if (ppos0 < 0)ppos0 += 12;
			if (ppos1 < 0)ppos1 += 12;
			if (ppos2 < 0)ppos2 += 12;


			newsym = topo[currenttopo][ppos2 + 12 * ppos1 + 12 * 12 * ppos0];

			tmpproj = asttopo[l];
			projpp2 = tmpproj % 12;
			tmpproj -= projpp2;
			tmpproj /= 12;
			projpp1 = tmpproj % 12;
			tmpproj -= projpp1;
			tmpproj /= 12;
			
			ppos0 += tmpproj;
			ppos0 %= 12;
			ppos1 += projpp1;
			ppos1 %= 12;
			ppos2 += projpp2;
			ppos2 %= 12;

			asttopo[l] = ppos2 + 12 * ppos1 + 12 * 12 * ppos0;



			dpos0 = fmod(dpos0, 1.0);
			if (dpos0 < 0) dpos0++;
			dpos0 -= 0.5;
			dpos1 = fmod(dpos1, 1.0);
			if (dpos1 < 0) dpos1++;
			dpos1 -= 0.5;
			dpos2 = fmod(dpos2, 1.0);
			if (dpos2 < 0) dpos2++;
			dpos2 -= 0.5;

			postmp[0] = dpos0;
			postmp[1] = dpos1;
			postmp[2] = dpos2;

			dpos0 = gsign[newsym][0] * postmp[gperm[newsym][0]];
			dpos1 = gsign[newsym][1] * postmp[gperm[newsym][1]];
			dpos2 = gsign[newsym][2] * postmp[gperm[newsym][2]];



			astpos0[l] = dpos0;
			astpos1[l] = dpos1;
			astpos2[l] = dpos2;


			nastv[0] = astv0[l];
			nastv[1] = astv1[l];
			nastv[2] = astv2[l];

			for (j = 0; j < 3; j++) nnastv[j] = gsign[newsym][j] * nastv[gperm[newsym][j]];

			astv0[l] = nnastv[0];
			astv1[l] = nnastv[1];
			astv2[l] = nnastv[2];
			
		}
	}
	
	for (i = 0; i < 3; i++)
	{
		if (lout[i])
		{

			lpos0[i] += lv0[i];
			lpos1[i] += lv1[i];
			lpos2[i] += lv2[i];

			if (wall[0])
			{
				tmpd = lpos2[i] + lsize - 0.5;
				if (tmpd > 0)
				{
					lpos2[i] -= tmpd;
					lv2[i] *= -1;
				}
			}
			if (wall[1])
			{
				tmpd = lpos0[i] + lsize - 0.5;
				if (tmpd > 0)
				{
					lpos0[i] -= tmpd;
					lv0[i] *= -1;
				}
			}
			if (wall[2])
			{
				tmpd = lpos1[i] + lsize - 0.5;
				if (tmpd > 0)
				{
					lpos1[i] -= tmpd;
					lv1[i] *= -1;
				}
			}
			if (wall[5])
			{
				tmpd = lpos2[i] - lsize + 0.5;
				if (tmpd < 0)
				{
					lpos2[i] -= tmpd;
					lv2[i] *= -1;
				}
			}
			if (wall[3])
			{
				tmpd = lpos0[i] - lsize + 0.5;
				if (tmpd < 0)
				{
					lpos0[i] -= tmpd;
					lv0[i] *= -1;
				}
			}
			if (wall[4])
			{
				tmpd = lpos1[i] - lsize + 0.5;
				if (tmpd < 0)
				{
					lpos1[i] -= tmpd;
					lv1[i] *= -1;
				}
			}


			dpos0 = lpos0[i] + 0.5;
			dpos1 = lpos1[i] + 0.5;
			dpos2 = lpos2[i] + 0.5;

			ppos0 = dpos0;
			ppos1 = dpos1;
			ppos2 = dpos2;

			if (dpos0 < 0) ppos0--;
			if (dpos1 < 0) ppos1--;
			if (dpos2 < 0) ppos2--;

			if (ppos0 != 0 || ppos1 != 0 || ppos2 != 0)
			{
				ppos0 %= 12;
				ppos1 %= 12;
				ppos2 %= 12;
				if (ppos0 < 0)ppos0 += 12;
				if (ppos1 < 0)ppos1 += 12;
				if (ppos2 < 0)ppos2 += 12;


				newsym = topo[currenttopo][ppos2 + 12 * ppos1 + 12 * 12 * ppos0];

				dpos0 = fmod(dpos0, 1.0);
				if (dpos0 < 0) dpos0++;
				dpos0 -= 0.5;
				dpos1 = fmod(dpos1, 1.0);
				if (dpos1 < 0) dpos1++;
				dpos1 -= 0.5;
				dpos2 = fmod(dpos2, 1.0);
				if (dpos2 < 0) dpos2++;
				dpos2 -= 0.5;

				postmp[0] = dpos0;
				postmp[1] = dpos1;
				postmp[2] = dpos2;

				dpos0 = gsign[newsym][0] * postmp[gperm[newsym][0]];
				dpos1 = gsign[newsym][1] * postmp[gperm[newsym][1]];
				dpos2 = gsign[newsym][2] * postmp[gperm[newsym][2]];



				lpos0[i] = dpos0;
				lpos1[i] = dpos1;
				lpos2[i] = dpos2;


				nastv[0] = lv0[i];
				nastv[1] = lv1[i];
				nastv[2] = lv2[i];

				for (j = 0; j < 3; j++) nnastv[j] = gsign[newsym][j] * nastv[gperm[newsym][j]];

				lv0[i] = nnastv[0];
				lv1[i] = nnastv[1];
				lv2[i] = nnastv[2];
			}
		}
	}


	for (l = 0; l < astnum[0]; l++)
	{
		for (j = l+1; j < astnum[0]; j++)
		{
			coord0 = astpos0[l] - astpos0[j];
			coord1 = astpos1[l] - astpos1[j];
			coord2 = astpos2[l] - astpos2[j];
			disto = sqrt(coord0 * coord0 + coord1 * coord1 + coord2 * coord2);
			if (disto < astrad[l] + astrad[j])
			{
				astv0[l] = coord0 * .001 / disto;
				astv1[l] = coord1 * .001 / disto;
				astv2[l] = coord2 * .001 / disto;

				astv0[j] = -coord0 * .001 / disto;
				astv1[j] = -coord1 * .001 / disto;
				astv2[j] = -coord2 * .001 / disto;
			}
		}
		for (j = astnum[0]; j < astnum[0] + astnumplus[0]; j++)
		{
			if (astplusidx[j] != l)
			{
				coord0 = astpos0[l] - astpos0[j];
				coord1 = astpos1[l] - astpos1[j];
				coord2 = astpos2[l] - astpos2[j];
				disto = sqrt(coord0 * coord0 + coord1 * coord1 + coord2 * coord2);
				if (disto < astrad[l] + astrad[j])
				{
					astv0[l] = coord0 * .001 / disto;
					astv1[l] = coord1 * .001 / disto;
					astv2[l] = coord2 * .001 / disto;
				}
			}
		}
	}
	
	for (l = 0; l < astnum[0]; l++)
	{
		for (i = -1; i < 2; i++)
		{
			for (j = -1; j < 2; j++)
			{
				for (k = -1; k < 2; k++)
				{
					if (i != 0 || j != 0 || k != 0)
					{
						ppos0 = i;
						ppos1 = j;
						ppos2 = k;

						postmp[0] = astpos0[l];
						postmp[1] = astpos1[l];
						postmp[2] = astpos2[l];

						if (ppos0 < 0)ppos0 += 12;
						if (ppos1 < 0)ppos1 += 12;
						if (ppos2 < 0)ppos2 += 12;

						newsym = gginv[topo[currenttopo][ppos2 + 12 * ppos1 + 12 * 12 * ppos0]];

						dpos0 = gsign[newsym][0] * postmp[gperm[newsym][0]];
						dpos1 = gsign[newsym][1] * postmp[gperm[newsym][1]];
						dpos2 = gsign[newsym][2] * postmp[gperm[newsym][2]];

						dpos0 += i;
						dpos1 += j;
						dpos2 += k;

						if ((dpos0 - astrad[l] < 0.5 && dpos0 - astrad[l]> -0.5) || (dpos0 + astrad[l] < 0.5 && dpos0 + astrad[l]> -0.5))
						{
							if ((dpos1 - astrad[l] < 0.5 && dpos1 - astrad[l]> -0.5) || (dpos1 + astrad[l] < 0.5 && dpos1 + astrad[l]> -0.5))
							{
								if ((dpos2 - astrad[l] < 0.5 && dpos2 - astrad[l]> -0.5) || (dpos2 + astrad[l] < 0.5 && dpos2 + astrad[l]> -0.5))
								{
									astrad[astnum[0] + astnumplustmp] = astrad[l];
									astpos0[astnum[0] + astnumplustmp] = dpos0;
									astpos1[astnum[0] + astnumplustmp] = dpos1;
									astpos2[astnum[0] + astnumplustmp] = dpos2;
									astplusidx[astnum[0] + astnumplustmp] = l;
									astsym[astnum[0] + astnumplustmp] = gginv[newsym];
									astnumplustmp++;
								}
							}
						}



					}
				}
			}
		}
	}
	
	ccpos0[0] = pos0 - 0.5;
	ccpos1[0] = pos1 - 0.5;
	ccpos2[0] = pos2 - 0.5;

	for (i = -1; i < 2; i++)
	{
		for (j = -1; j < 2; j++)
		{
			for (k = -1; k < 2; k++)
			{
				if (i != 0 || j != 0 || k != 0)
				{
					ppos0 = i;
					ppos1 = j;
					ppos2 = k;

					postmp[0] = ccpos0[0];
					postmp[1] = ccpos1[0];
					postmp[2] = ccpos2[0];

					if (ppos0 < 0)ppos0 += 12;
					if (ppos1 < 0)ppos1 += 12;
					if (ppos2 < 0)ppos2 += 12;

					newsym = gginv[topo[currenttopo][ppos2 + 12 * ppos1 + 12*12*ppos0]];

				

					dpos0 = gsign[newsym][0] * postmp[gperm[newsym][0]];
					dpos1 = gsign[newsym][1] * postmp[gperm[newsym][1]];
					dpos2 = gsign[newsym][2] * postmp[gperm[newsym][2]];

					dpos0 += i;
					dpos1 += j;
					dpos2 += k;

					if ((dpos0 - csize < 0.5 && dpos0 - csize> -0.5) || (dpos0 + csize < 0.5 && dpos0 + csize> -0.5))
					{
						if ((dpos1 - csize < 0.5 && dpos1 - csize> -0.5) || (dpos1 + csize < 0.5 && dpos1 + csize> -0.5))
						{
							if ((dpos2 - csize < 0.5 && dpos2 - csize> -0.5) || (dpos2 + csize < 0.5 && dpos2 + csize> -0.5))
							{
								ccpos0[cnumplustmp] = dpos0;
								ccpos1[cnumplustmp] = dpos1;
								ccpos2[cnumplustmp] = dpos2;
								ccsym[cnumplustmp] = gginv[newsym];
								cnumplustmp++;
							}
						}
					}



				}
			}
		}
	}

	for (ii = 0; ii < 3; ii++)
	{
		if (lout[ii])
		{
			for (l = 0; l < astnum[0] + astnumplus[0]; l++)
			{
				coord0 = astpos0[l] - lpos0[ii];
				coord1 = astpos1[l] - lpos1[ii];
				coord2 = astpos2[l] - lpos2[ii];
				disto = sqrt(coord0 * coord0 + coord1 * coord1 + coord2 * coord2);
				if (disto < astrad[l] + lsize)
				{
					if (l >= astnum[0]) tmpidx = astplusidx[l];
					else tmpidx = l;

					if (astrad[tmpidx] < 2 * lsize)
					{
						astrad[tmpidx] = astrad[astnum[0] - 1];
						astpos0[tmpidx] = astpos0[astnum[0] - 1];
						astpos1[tmpidx] = astpos1[astnum[0] - 1];
						astpos2[tmpidx] = astpos2[astnum[0] - 1];
						astv0[tmpidx] = astv0[astnum[0] - 1];
						astv1[tmpidx] = astv1[astnum[0] - 1];
						astv2[tmpidx] = astv2[astnum[0] - 1];
						asttopo[tmpidx] = asttopo[astnum[0] - 1];
						
						for (k = 0; k < 9; k++)
						{
							tmpmat = astmat[tmpidx][k];
							astmat[tmpidx][k] = astmat[astnum[0]-1][k];
							astmat[astnum[0] - 1][k] = tmpmat;

							tmpmat = astmatv[tmpidx][k];
							astmatv[tmpidx][k] = astmatv[astnum[0] - 1][k];
							astmatv[astnum[0] - 1][k] = tmpmat;
						}
						
						
						astnum[0]--;

						lout[ii] = false;
				

						l = 2000;
					}
					else
					{
						theta = fmod((double)nbframe, 2 * M_PI);
						phi = fmod((double)(nbframe * nbframe), M_PI);

						tmp0 = sin(phi) * cos(theta);
						tmp1 = sin(phi) * sin(theta);
						tmp2 = cos(phi);

						astrad[tmpidx] /= 2;
						astrad[astnum[0]] = astrad[tmpidx];

						astpos0[astnum[0]] = astpos0[tmpidx] + tmp0 * astrad[tmpidx];
						astpos1[astnum[0]] = astpos1[tmpidx] + tmp1 * astrad[tmpidx];
						astpos2[astnum[0]] = astpos2[tmpidx] + tmp2 * astrad[tmpidx];

						astpos0[tmpidx] -= tmp0 * astrad[tmpidx];
						astpos1[tmpidx] -= tmp1 * astrad[tmpidx];
						astpos2[tmpidx] -= tmp2 * astrad[tmpidx];


						astv0[tmpidx] = -0.001 * tmp0;
						astv1[tmpidx] = -0.001 * tmp1;
						astv2[tmpidx] = -0.001 * tmp2;

						astv0[astnum[0]] = 0.001 * tmp0;
						astv1[astnum[0]] = 0.001 * tmp1;
						astv2[astnum[0]] = 0.001 * tmp2;

						asttopo[tmpidx] = 0;
						asttopo[astnum[0]] = 0;

						dpos0 = astpos0[tmpidx] + 0.5;
						dpos1 = astpos1[tmpidx] + 0.5;
						dpos2 = astpos2[tmpidx] + 0.5;

						ppos0 = dpos0;
						ppos1 = dpos1;
						ppos2 = dpos2;

						if (dpos0 < 0) ppos0--;
						if (dpos1 < 0) ppos1--;
						if (dpos2 < 0) ppos2--;

						if (ppos0 != 0 || ppos1 != 0 || ppos2 != 0)
						{
							ppos0 %= 12;
							ppos1 %= 12;
							ppos2 %= 12;
							if (ppos0 < 0)ppos0 += 12;
							if (ppos1 < 0)ppos1 += 12;
							if (ppos2 < 0)ppos2 += 12;


							newsym = topo[currenttopo][ppos2 + 12 * ppos1 + 12 * 12 * ppos0];

							tmpproj = asttopo[tmpidx];
							projpp2 = tmpproj % 12;
							tmpproj -= projpp2;
							tmpproj /= 12;
							projpp1 = tmpproj % 12;
							tmpproj -= projpp1;
							tmpproj /= 12;

							ppos0 += tmpproj;
							ppos0 %= 12;
							ppos1 += projpp1;
							ppos1 %= 12;
							ppos2 += projpp2;
							ppos2 %= 12;

							asttopo[tmpidx] = ppos2 + 12 * ppos1 + 12 * 12 * ppos0;

							dpos0 = fmod(dpos0, 1.0);
							if (dpos0 < 0) dpos0++;
							dpos0 -= 0.5;
							dpos1 = fmod(dpos1, 1.0);
							if (dpos1 < 0) dpos1++;
							dpos1 -= 0.5;
							dpos2 = fmod(dpos2, 1.0);
							if (dpos2 < 0) dpos2++;
							dpos2 -= 0.5;

							postmp[0] = dpos0;
							postmp[1] = dpos1;
							postmp[2] = dpos2;

							dpos0 = gsign[newsym][0] * postmp[gperm[newsym][0]];
							dpos1 = gsign[newsym][1] * postmp[gperm[newsym][1]];
							dpos2 = gsign[newsym][2] * postmp[gperm[newsym][2]];



							astpos0[tmpidx] = dpos0;
							astpos1[tmpidx] = dpos1;
							astpos2[tmpidx] = dpos2;


							nastv[0] = astv0[tmpidx];
							nastv[1] = astv1[tmpidx];
							nastv[2] = astv2[tmpidx];

							for (j = 0; j < 3; j++) nnastv[j] = gsign[newsym][j] * nastv[gperm[newsym][j]];

							astv0[tmpidx] = nnastv[0];
							astv1[tmpidx] = nnastv[1];
							astv2[tmpidx] = nnastv[2];
						}

						dpos0 = astpos0[astnum[0]] + 0.5;
						dpos1 = astpos1[astnum[0]] + 0.5;
						dpos2 = astpos2[astnum[0]] + 0.5;

						ppos0 = dpos0;
						ppos1 = dpos1;
						ppos2 = dpos2;

						if (dpos0 < 0) ppos0--;
						if (dpos1 < 0) ppos1--;
						if (dpos2 < 0) ppos2--;

						if (ppos0 != 0 || ppos1 != 0 || ppos2 != 0)
						{
							ppos0 %= 12;
							ppos1 %= 12;
							ppos2 %= 12;
							if (ppos0 < 0)ppos0 += 12;
							if (ppos1 < 0)ppos1 += 12;
							if (ppos2 < 0)ppos2 += 12;


							newsym = topo[currenttopo][ppos2 + 12 * ppos1 + 12 * 12 * ppos0];

							tmpproj = asttopo[astnum[0]];
							projpp2 = tmpproj % 12;
							tmpproj -= projpp2;
							tmpproj /= 12;
							projpp1 = tmpproj % 12;
							tmpproj -= projpp1;
							tmpproj /= 12;

							ppos0 += tmpproj;
							ppos0 %= 12;
							ppos1 += projpp1;
							ppos1 %= 12;
							ppos2 += projpp2;
							ppos2 %= 12;

							asttopo[astnum[0]] = ppos2 + 12 * ppos1 + 12 * 12 * ppos0;

							dpos0 = fmod(dpos0, 1.0);
							if (dpos0 < 0) dpos0++;
							dpos0 -= 0.5;
							dpos1 = fmod(dpos1, 1.0);
							if (dpos1 < 0) dpos1++;
							dpos1 -= 0.5;
							dpos2 = fmod(dpos2, 1.0);
							if (dpos2 < 0) dpos2++;
							dpos2 -= 0.5;

							postmp[0] = dpos0;
							postmp[1] = dpos1;
							postmp[2] = dpos2;

							dpos0 = gsign[newsym][0] * postmp[gperm[newsym][0]];
							dpos1 = gsign[newsym][1] * postmp[gperm[newsym][1]];
							dpos2 = gsign[newsym][2] * postmp[gperm[newsym][2]];



							astpos0[astnum[0]] = dpos0;
							astpos1[astnum[0]] = dpos1;
							astpos2[astnum[0]] = dpos2;


							nastv[0] = astv0[astnum[0]];
							nastv[1] = astv1[astnum[0]];
							nastv[2] = astv2[astnum[0]];

							for (j = 0; j < 3; j++) nnastv[j] = gsign[newsym][j] * nastv[gperm[newsym][j]];

							astv0[astnum[0]] = nnastv[0];
							astv1[astnum[0]] = nnastv[1];
							astv2[astnum[0]] = nnastv[2];
						}
						astnum[0]++;

						astnumplustmp = 0;
						for (l = 0; l < astnum[0]; l++)
						{
							for (i = -1; i < 2; i++)
							{
								for (j = -1; j < 2; j++)
								{
									for (k = -1; k < 2; k++)
									{
										if (i != 0 || j != 0 || k != 0)
										{
											ppos0 = i;
											ppos1 = j;
											ppos2 = k;

											postmp[0] = astpos0[l];
											postmp[1] = astpos1[l];
											postmp[2] = astpos2[l];

											if (ppos0 < 0)ppos0 += 12;
											if (ppos1 < 0)ppos1 += 12;
											if (ppos2 < 0)ppos2 += 12;

											newsym = gginv[topo[currenttopo][ppos2 + 12 * ppos1 + 12 * 12 * ppos0]];

											dpos0 = gsign[newsym][0] * postmp[gperm[newsym][0]];
											dpos1 = gsign[newsym][1] * postmp[gperm[newsym][1]];
											dpos2 = gsign[newsym][2] * postmp[gperm[newsym][2]];

											dpos0 += i;
											dpos1 += j;
											dpos2 += k;

											if ((dpos0 - astrad[l] < 0.5 && dpos0 - astrad[l]> -0.5) || (dpos0 + astrad[l] < 0.5 && dpos0 + astrad[l]> -0.5))
											{
												if ((dpos1 - astrad[l] < 0.5 && dpos1 - astrad[l]> -0.5) || (dpos1 + astrad[l] < 0.5 && dpos1 + astrad[l]> -0.5))
												{
													if ((dpos2 - astrad[l] < 0.5 && dpos2 - astrad[l]> -0.5) || (dpos2 + astrad[l] < 0.5 && dpos2 + astrad[l]> -0.5))
													{
														astrad[astnum[0] + astnumplustmp] = astrad[l];
														astpos0[astnum[0] + astnumplustmp] = dpos0;
														astpos1[astnum[0] + astnumplustmp] = dpos1;
														astpos2[astnum[0] + astnumplustmp] = dpos2;
														astplusidx[astnum[0] + astnumplustmp] = l;
														astsym[astnum[0] + astnumplustmp] = gginv[newsym];
														astnumplustmp++;
													}
												}
											}



										}
									}
								}
							}
						}


						lout[ii] = false;


						l = 2000;
					}
				}
			}
		}
	}


	for (ii = 0; ii < 3; ii++)
	{
		if (lout[ii])
		{
			for (i = -1; i < 2; i++)
			{
				for (j = -1; j < 2; j++)
				{
					for (k = -1; k < 2; k++)
					{
						if (i != 0 || j != 0 || k != 0)
						{
							ppos0 = i;
							ppos1 = j;
							ppos2 = k;

							postmp[0] = lpos0[ii];
							postmp[1] = lpos1[ii];
							postmp[2] = lpos2[ii];

							if (ppos0 < 0)ppos0 += 12;
							if (ppos1 < 0)ppos1 += 12;
							if (ppos2 < 0)ppos2 += 12;

							newsym = gginv[topo[currenttopo][ppos2 + 12 * ppos1 + 12 * 12 * ppos0]];

							dpos0 = gsign[newsym][0] * postmp[gperm[newsym][0]];
							dpos1 = gsign[newsym][1] * postmp[gperm[newsym][1]];
							dpos2 = gsign[newsym][2] * postmp[gperm[newsym][2]];

							dpos0 += i;
							dpos1 += j;
							dpos2 += k;

							if ((dpos0 - lsize < 0.5 && dpos0 - lsize> -0.5) || (dpos0 + lsize < 0.5 && dpos0 + lsize> -0.5))
							{
								if ((dpos1 - lsize < 0.5 && dpos1 - lsize> -0.5) || (dpos1 + lsize < 0.5 && dpos1 + lsize> -0.5))
								{
									if ((dpos2 - lsize < 0.5 && dpos2 - lsize> -0.5) || (dpos2 + lsize < 0.5 && dpos2 + lsize> -0.5))
									{
										lpos0[3+lnumplustmp] = dpos0;
										lpos1[3+lnumplustmp] = dpos1;
										lpos2[3+lnumplustmp] = dpos2;
										lidx[3 + lnumplustmp] = ii;
										lnumplustmp++;
									}
								}
							}



						}
					}
				}
			}
		}
	}

	for (l = 0; l < astnum[0] + astnumplus[0]; l++)
	{
			coord0 = astpos0[l] - ccpos0[0];
			coord1 = astpos1[l] - ccpos1[0];
			coord2 = astpos2[l] - ccpos2[0];
			disto = sqrt(coord0 * coord0 + coord1 * coord1 + coord2 * coord2);
			if (disto < astrad[l] + csize)
			{
				rip[0] = true;
			}
			
			haz[0] = 1- (disto- astrad[l]-csize) / ( 2*csize);
			if (haz[0] < 0) haz[0] = 0;
			if (haz[0] > hazmax) hazmax = haz[0];
	}

	for (l = 0; l < 3; l++)
	{
		if (lout[l] && nbframe - loutf[l] > 8)
		{
			coord0 = lpos0[l] - ccpos0[0];
			coord1 = lpos1[l] - ccpos1[0];
			coord2 = lpos2[l] - ccpos2[0];
			disto= sqrt(coord0 * coord0 + coord1 * coord1 + coord2 * coord2);
			if (disto < lsize + csize)
			{
				rip[0] = true;
			}

			haz[0] = 1 - (disto - lsize - csize) / (2 * csize);
			if (haz[0] < 0) haz[0] = 0;
			if (haz[0] > hazmax) hazmax = haz[0];
		}
	}

	
	for (l = 0; l < lnumplus[0]; l++)
	{
		if (nbframe - loutf[lidx[3 + l]] > 8)
		{
			coord0 = lpos0[3 + l] - ccpos0[0];
			coord1 = lpos1[3 + l] - ccpos1[0];
			coord2 = lpos2[3 + l] - ccpos2[0];
			disto = sqrt(coord0 * coord0 + coord1 * coord1 + coord2 * coord2);
			if (disto < lsize + csize)
			{
				rip[0] = true;
			}

			haz[0] = 1 - (disto - lsize - csize) / (2 * csize);
			if (haz[0] < 0) haz[0] = 0;
			if (haz[0] > hazmax) hazmax = haz[0];
		}
	}

	for (l = 0; l < astnum[0]; l++) { astplusidx[l] = l; astsym[l] = 0; }

	haz[0] = hazmax;
	lnumplus[0] = lnumplustmp;
	astnumplus[0] = astnumplustmp;
	cnumplus[0] = cnumplustmp;
}

__global__ void addKernel(uint8_t * buffer, int** gsyms, int** ggstruct, int** gperm, int** gsign, double* vecl, double pos0, double pos1, double pos2, double vec0, double vec1, double vec2, double addy0, double addy1, double addy2, double addz0, double addz1, double addz2, int rep0, int rep1, int rep2, int rep3, int rep4, int rep5, int* astnum, int* astnumplus, double* astrad, double* astpos0, double* astpos1, double* astpos2, int currenttopo, double* ccpos0, double* ccpos1, double* ccpos2, int* cnumplus, double* lpos0, double* lpos1, double* lpos2, int* lnumplus,bool* rip, int nbframe, double* haz, int* lidx, bool* lout, double* seedist, bool w0, bool w1, bool w2, bool w3, bool w4, bool w5, int* loutf, double m0, double m1, double m2, double m3, double m4, double m5, double m6, double m7, double m8, int* ccsym, int* astsym, double** astmat, int* astplusidx, int* asttopo, int** topo)
	{
	
	double csize = 0.05;
	double lsize = 0.01;
	int i,l;
	double vecn[3]{};
	double nvecn[3]{};
	double coll[3]{};
	double lastcollmin=0;
	double inv[3]{};
	int min;
	double lfac;
	double lcor[3]{};

	double lcol = 0;
	double lcormax;
	int lcorr, lcorg, lcorb;

	double conttmp[3]{};

	int rnbwv;
	int colr, colg, col;

	int tmp = blockIdx.x * blockDim.x + threadIdx.x;
	int tmpx = tmp % 1920;
	int tmpy = (tmp - tmpx) / 1920;
	int ngt[3]{};
	int ngt2[3]{};
	int newsym2;

	int reps[6]{};
	int rand = tmp;
	int noise;
	int wh;
	int uv;
	int coordx, coordy, coord;

	double dark;

	int astcolidx;

	double qa, qb, qc, discr;
	double t1;

	double astcolmin;
	double cpos[3]{};
	int currentsym = 0;
	int newsym;

	double ncpos[3]{};
	double lfe[3]{};
	bool wall[6]{};

	int ccolidx;
	double cont[3]{};
	double ncont0, ncont1, ncont2;
	double u, v;
	double vf;
	int ccor0, ccor1, ccor2;

	wall[0] = w0;
	wall[1] = w1;
	wall[2] = w2;
	wall[3] = w3;
	wall[4] = w4;
	wall[5] = w5;

	for (l = 0; l < 10; l++) rand = (60493 * rand + 11) % 479001599;
	rand += nbframe;
	rand %= 479001599;
	for (l = 0; l < 10; l++) rand = (60493 * rand + 11) % 479001599;

	if (astnum[0] == 0)
	{
		if ((tmpx >= 240 && tmpx <= 240 + 4 * 144 && tmpy >= 135 && tmpy <= 135 + 162) ||
			(tmpx >= 240 && tmpx <= 240 + 144 && tmpy >= 135 && tmpy <= 135 + 5 * 162) ||
			(tmpx >= 240 && tmpx <= 240 + 4 * 144 && tmpy >= 135 + 4 * 162 && tmpy <= 135 + 5 * 162) ||
			(tmpx >= 240 + 3 * 144 && tmpx <= 240 + 4 * 144 && tmpy >= 135 + 2 * 162 && tmpy <= 135 + 5 * 162) ||
			(tmpx >= 240 + 2 * 144 && tmpx <= 240 + 3 * 144 && tmpy >= 135 + 2 * 162 && tmpy <= 135 + 3 * 162) ||

			(tmpx >= 240 + 864 && tmpx <= 240 + 4 * 144 + 864 && tmpy >= 135 && tmpy <= 135 + 162) ||
			(tmpx >= 240 + 864 && tmpx <= 240 + 144 + 864 && tmpy >= 135 && tmpy <= 135 + 5 * 162) ||
			(tmpx >= 240 + 864 && tmpx <= 240 + 4 * 144 + 864 && tmpy >= 135 + 4 * 162 && tmpy <= 135 + 5 * 162) ||
			(tmpx >= 240 + 3 * 144 + 864 && tmpx <= 240 + 4 * 144 + 864 && tmpy >= 135 + 2 * 162 && tmpy <= 135 + 5 * 162) ||
			(tmpx >= 240 + 2 * 144 + 864 && tmpx <= 240 + 3 * 144 + 864 && tmpy >= 135 + 2 * 162 && tmpy <= 135 + 3 * 162))
		{
			rnbwv = rnbw(nbframe);

			colr = rnbwv % 256;
			rnbwv -= colr;
			rnbwv /= 256;
			colg = rnbwv % 256;
			rnbwv -= colg;
			rnbwv /= 256;
			col = rnbwv % 256;

			buffer[4 * tmp] = colr;
			buffer[4 * tmp + 1] = colg;
			buffer[4 * tmp + 2] = col;

			return;
		}

		
	}


	if (rand % 2 == 0) noise = 0;
	else noise = 255;

	if (rip[0])
	{
		if (rand % 2 == 0)
		{
			buffer[4 * tmp] = noise;
			buffer[4 * tmp + 1] = noise;
			buffer[4 * tmp + 2] = noise;
		}
		else
		{
			buffer[4 * tmp] = noise;
			buffer[4 * tmp + 1] = noise;
			buffer[4 * tmp + 2] = noise;
		}

		return;
	}

	for (i = 0; i < 3; i++)
	{
		if (lout[i])
		{
			if (nbframe - loutf[i] > 150) lfe[i] = 4.0 - (nbframe - loutf[i]) / 50.0;
			else lfe[i] = 1.0;
		}
	}


	vecn[0] = vec0 + tmpx * addy0 + tmpy * addz0;
	vecn[1] = vec1 + tmpx * addy1 + tmpy * addz1;
	vecn[2] = vec2 + tmpx * addy2 + tmpy * addz2;

	vecn[0] *= vecl[tmp];
	vecn[1] *= vecl[tmp];
	vecn[2] *= vecl[tmp];

	reps[0] = rep0;
	reps[1] = rep1;
	reps[2] = rep2;
	reps[3] = rep3;
	reps[4] = rep4;
	reps[5] = rep5;

	cpos[0] = pos0 - 0.5;
	cpos[1] = pos1 - 0.5;
	cpos[2] = pos2 - 0.5;
	astcolmin = 2;
	qa = vecn[0] * vecn[0] + vecn[1] * vecn[1] + vecn[2] * vecn[2];
	for (i = 0; i < astnum[0] + astnumplus[0]; i++)
	{
		
		qb = 2 * (vecn[0] * (cpos[0] - astpos0[i]) + vecn[1] * (cpos[1] - astpos1[i]) + vecn[2] * (cpos[2] - astpos2[i]));
		qc = (cpos[0] - astpos0[i]) * (cpos[0] - astpos0[i]) + (cpos[1] - astpos1[i]) * (cpos[1] - astpos1[i]) + (cpos[2] - astpos2[i]) * (cpos[2] - astpos2[i]) - astrad[i] * astrad[i];

		discr = qb * qb - 4 * qa * qc;

		if (discr > 0)
		{
			t1 = ((-1.0) * qb - sqrt(discr)) / (2.0 * qa);
			if (t1 > 0 && t1 < astcolmin)
			{
				astcolmin = t1;
				astcolidx = i;
			}
		}
	}

	for (i = 0; i < cnumplus[0]; i++)
	{
		qb = 2 * (vecn[0] * (cpos[0] - ccpos0[i]) + vecn[1] * (cpos[1] - ccpos1[i]) + vecn[2] * (cpos[2] - ccpos2[i]));
		qc = (cpos[0] - ccpos0[i]) * (cpos[0] - ccpos0[i]) + (cpos[1] - ccpos1[i]) * (cpos[1] - ccpos1[i]) + (cpos[2] - ccpos2[i]) * (cpos[2] - ccpos2[i]) - csize*csize;

		discr = qb * qb - 4 * qa * qc;

		if (discr > 0)
		{
			t1 = ((-1.0) * qb - sqrt(discr)) / (2.0 * qa);
			if (t1 > 0 && t1 < astcolmin)
			{
				astcolmin = t1;
				astcolidx = -1;
				ccolidx = i;
			}
		}
	}
	
	
	ngt[0] = (2 * signbit(vecn[0]) - 1) * -1;
	ngt2[0] = (signbit(vecn[0]) + 1) % 2;
	inv[0] = 1 / vecn[0];
	coll[0] = inv[0] * (ngt2[0] - pos0);
	inv[0] *= ngt[0];

	ngt[1] = (2 * signbit(vecn[1]) - 1) * -1;
	ngt2[1] = (signbit(vecn[1]) + 1) % 2;
	inv[1] = 1 / vecn[1];
	coll[1] = inv[1] * (ngt2[1] - pos1);
	inv[1] *= ngt[1];

	ngt[2] = (2 * signbit(vecn[2]) - 1) * -1;
	ngt2[2] = (signbit(vecn[2]) + 1) % 2;
	inv[2] = 1 / vecn[2];
	coll[2] = inv[2] * (ngt2[2] - pos2);
	inv[2] *= ngt[2];

	if (coll[0] < coll[1]) min = 0;
	else min = 1;
	if (coll[2] < coll[min]) min = 2;

	lcor[0] = 0;
	lcor[1] = 0;
	lcor[2] = 0;

	for (i = 0; i < 3; i++)
	{
		if (lout[i])
		{
			qb = 2 * (vecn[0] * (cpos[0] - lpos0[i]) + vecn[1] * (cpos[1] - lpos1[i]) + vecn[2] * (cpos[2] - lpos2[i]));
			qc = (cpos[0] - lpos0[i]) * (cpos[0] - lpos0[i]) + (cpos[1] - lpos1[i]) * (cpos[1] - lpos1[i]) + (cpos[2] - lpos2[i]) * (cpos[2] - lpos2[i]) - lsize * lsize;

			discr = qb * qb - 4 * qa * qc;

			if (discr > 0)
			{
				t1 = ((-1.0) * qb - sqrt(discr)) / (2.0 * qa);
				if (t1 > 0 && t1 < astcolmin && t1<coll[min])
				{
					lcor[i] += ((-1.0) * qb + sqrt(discr)) / (2.0 * qa) - t1;
				}
			}
		}
	}

	for (i = 0; i < lnumplus[0]; i++)
	{
		qb = 2 * (vecn[0] * (cpos[0] - lpos0[3 + i]) + vecn[1] * (cpos[1] - lpos1[3 + i]) + vecn[2] * (cpos[2] - lpos2[3 + i]));
		qc = (cpos[0] - lpos0[3 + i]) * (cpos[0] - lpos0[3 + i]) + (cpos[1] - lpos1[3 + i]) * (cpos[1] - lpos1[3 + i]) + (cpos[2] - lpos2[3 + i]) * (cpos[2] - lpos2[3 + i]) - lsize * lsize;

		discr = qb * qb - 4 * qa * qc;

		if (discr > 0)
		{
			t1 = ((-1.0) * qb - sqrt(discr)) / (2.0 * qa);
			if (t1 > 0 && t1 < astcolmin && t1 < coll[min])
			{
				lcor[lidx[3 + i]] += ((-1.0) * qb + sqrt(discr)) / (2.0 * qa) - t1;
			}
		}
	}
	
	if (astcolmin < coll[min])
	{
		lcor[0] *= lfe[0];
		lcor[1] *= lfe[1];
		lcor[2] *= lfe[2];

		lcol = lcor[0] + lcor[1] + lcor[2];
		dark = (-1.0 / seedist[0]) * (astcolmin) + 1.0;
		if (dark < 0) dark = 0.0;
		lfac = (1.0 / (4 * lsize * lsize)) * lcol * lcol;
		if (lfac > 1.0) lfac = 1.0;

		if (lcor[0] > lcor[1]) lcormax = lcor[0];
		else lcormax = lcor[1];
		if (lcor[2] > lcormax) lcormax = lcor[2];

		lcorr = 255.0*lcor[0] / lcormax;
		lcorg = 255.0 * lcor[1] / lcormax;
		lcorb = 255.0 * lcor[2] / lcormax;
	
		if (astcolidx == -1)
		{

			newsym = ccsym[ccolidx];

			cont[0] = (cpos[0] + astcolmin * vecn[0]) - ccpos0[ccolidx];
			cont[1] = (cpos[1] + astcolmin * vecn[1]) - ccpos1[ccolidx];
			cont[2] = (cpos[2] + astcolmin * vecn[2]) - ccpos2[ccolidx];

			conttmp[0] = gsign[newsym][0] * cont[gperm[newsym][0]];
			conttmp[1] = gsign[newsym][1] * cont[gperm[newsym][1]];
			conttmp[2] = gsign[newsym][2] * cont[gperm[newsym][2]];

			ncont0 = m0 * conttmp[0]+ m1 * conttmp[1] + m2 * conttmp[2];
			ncont1 = m3 * conttmp[0] + m4 * conttmp[1] + m5 * conttmp[2];
			ncont2 = m6 * conttmp[0] + m7 * conttmp[1] + m8 * conttmp[2];

			ncont0 /= csize;
			ncont1 /= csize;
			ncont2 /= csize;

			u = (0.5 + atan2(ncont1, ncont0) / (2.0 * M_PI));
			v = (0.5 + asin(ncont2) / M_PI);

			rnbwv = rnbw2(u);
			colr = rnbwv % 256;
			rnbwv -= colr;
			rnbwv /= 256;
			colg = rnbwv % 256;
			rnbwv -= colg;
			rnbwv /= 256;
			col = rnbwv % 256;

			if (v > 0.5)
			{
				vf = 2.0 * v - 1.0;

				ccor0 = (1 - vf) * colr + vf * 255.0;
				ccor1 = (1 - vf) * colg + vf * 255.0;
				ccor2 = (1 - vf) * col + vf * 255.0;
			}
			else
			{
				vf = 1.0 - 2.0 * v;

				ccor0 = (1 - vf) * colr;
				ccor1 = (1 - vf) * colg;
				ccor2 = (1 - vf) * col;
			}

			if ((u - 0.5) * (u - 0.5) + (v - 0.5) * (v - 0.5) < 0.0035)
			{
				ccor0 = 128;
				ccor1 = 128;
				ccor2 = 128;
			}
			if ((u - 0.5) * (u - 0.5) + (v - 0.5) * (v - 0.5) < 0.0025)
			{
				ccor0 = 0;
				ccor1 = 0;
				ccor2 = 0;
			}


			buffer[4 * tmp] = haz[0] * noise + (1 - haz[0]) * ((1 - lfac) * dark * (ccor0)+lfac * lcorr);
			buffer[4 * tmp + 1] = haz[0] * noise + (1 - haz[0]) * ((1 - lfac) * dark * (ccor1)+lfac * lcorg);
			buffer[4 * tmp + 2] = haz[0] * noise + (1 - haz[0]) * ((1 - lfac) * dark * (ccor2)+lfac * lcorb);
		}
		else {
			newsym = astsym[astcolidx];
			newsym2 = topo[currenttopo][asttopo[astplusidx[astcolidx]]];

			cont[0] = (cpos[0] + astcolmin * vecn[0]) - astpos0[astcolidx];
			cont[1] = (cpos[1] + astcolmin * vecn[1]) - astpos1[astcolidx];
			cont[2] = (cpos[2] + astcolmin * vecn[2]) - astpos2[astcolidx];

			conttmp[0] = gsign[newsym][0] * cont[gperm[newsym][0]];
			conttmp[1] = gsign[newsym][1] * cont[gperm[newsym][1]];
			conttmp[2] = gsign[newsym][2] * cont[gperm[newsym][2]];

			cont[0] = gsign[newsym2][0] * conttmp[gperm[newsym2][0]];
			cont[1] = gsign[newsym2][1] * conttmp[gperm[newsym2][1]];
			cont[2] = gsign[newsym2][2] * conttmp[gperm[newsym2][2]];


			ncont0 = astmat[astplusidx[astcolidx]][0] * cont[0] + astmat[astplusidx[astcolidx]][1] * cont[1] + astmat[astplusidx[astcolidx]][2] * cont[2];
			ncont1 = astmat[astplusidx[astcolidx]][3] * cont[0] + astmat[astplusidx[astcolidx]][4] * cont[1] + astmat[astplusidx[astcolidx]][5] * cont[2];
			ncont2 = astmat[astplusidx[astcolidx]][6] * cont[0]+ astmat[astplusidx[astcolidx]][7] * cont[1] + astmat[astplusidx[astcolidx]][8] * cont[2];

			ncont0 /= astrad[astplusidx[astcolidx]];
			ncont1 /= astrad[astplusidx[astcolidx]];
			ncont2 /= astrad[astplusidx[astcolidx]];

			u = 100.0*(0.5 + atan2(ncont1, ncont0) / (2.0 * M_PI));
			v = 100.0*(0.5 + asin(ncont2) / M_PI);
			
			uv = (int)u + 100 * (int)v;

			for (l = 0; l < 10; l++) uv = (60493 * uv + 11) % 479001599;

			uv %= 30;
			uv += 128;

			buffer[4 * tmp] = haz[0] * noise + (1 - haz[0]) * ((1 - lfac) * dark * uv + lfac * lcorr);
			buffer[4 * tmp + 1] = haz[0] * noise + (1 - haz[0]) * ((1 - lfac) * dark * uv + lfac * lcorg);
			buffer[4 * tmp + 2] = haz[0] * noise + (1 - haz[0]) * ((1 - lfac) * dark * uv + lfac * lcorb);
		}

		return;
	}

	

	while (coll[min] < seedist[0])
	{
		
		

		cpos[0] = pos0 + vecn[0] * coll[min];
		cpos[1] = pos1 + vecn[1] * coll[min];
		cpos[2] = pos2 + vecn[2] * coll[min];

		if (min == 0)
		{
			if (vecn[0] < 0)
			{
				wh = gsyms[currentsym][3];
				newsym = ggstruct[currentsym][reps[wh]];
				cpos[0] = 0.5;
				cpos[1] = fmod(cpos[1], 1.0);
				if (cpos[1] < 0)cpos[1]++;
				cpos[1] -= 0.5;
				cpos[2] = fmod(cpos[2], 1.0);
				if (cpos[2] < 0)cpos[2]++;
				cpos[2] -= 0.5;
				
			}
			else
			{
				wh = gsyms[currentsym][1];
				newsym = ggstruct[currentsym][reps[wh]];
				cpos[0] = -0.5;
				cpos[1] = fmod(cpos[1], 1.0);
				if (cpos[1] < 0)cpos[1]++;
				cpos[1] -= 0.5;
				cpos[2] = fmod(cpos[2], 1.0);
				if (cpos[2] < 0)cpos[2]++;
				cpos[2] -= 0.5;
				
			}
		}
		else if (min == 1)
		{
			if (vecn[1] < 0)
			{
				wh = gsyms[currentsym][4];
				newsym = ggstruct[currentsym][reps[wh]];
				cpos[0] = fmod(cpos[0], 1.0);
				if (cpos[0] < 0)cpos[0]++;
				cpos[0] -= 0.5;
				cpos[1] = 0.5;
				cpos[2] = fmod(cpos[2], 1.0);
				if (cpos[2] < 0)cpos[2]++;
				cpos[2] -= 0.5;
				

			}
			else
			{
				wh = gsyms[currentsym][2];
				newsym = ggstruct[currentsym][reps[wh]];
				cpos[0] = fmod(cpos[0], 1.0);
				if (cpos[0] < 0)cpos[0]++;
				cpos[0] -= 0.5;
				cpos[1] = -0.5;
				cpos[2] = fmod(cpos[2], 1.0);
				if (cpos[2] < 0)cpos[2]++;
				cpos[2] -= 0.5;
				
			}
		}
		else 
		{
			if (vecn[2] < 0)
			{
				wh = gsyms[currentsym][5];
				newsym = ggstruct[currentsym][reps[wh]];
				cpos[0] = fmod(cpos[0], 1.0);
				if (cpos[0] < 0)cpos[0]++;
				cpos[0] -= 0.5;
				cpos[1] = fmod(cpos[1], 1.0);
				if (cpos[1] < 0)cpos[1]++;
				cpos[1] -= 0.5;
				cpos[2] = 0.5;
				
			}
			else
			{
				wh = gsyms[currentsym][0];
				newsym = ggstruct[currentsym][reps[wh]];
				cpos[0] = fmod(cpos[0], 1.0);
				if (cpos[0] < 0)cpos[0]++;
				cpos[0] -= 0.5;
				cpos[1] = fmod(cpos[1], 1.0);
				if (cpos[1] < 0)cpos[1]++;
				cpos[1] -= 0.5;
				cpos[2] = -0.5;
				
			}
		}

		ncpos[0] = gsign[newsym][0] * cpos[gperm[newsym][0]];
		ncpos[1] = gsign[newsym][1] * cpos[gperm[newsym][1]];
		ncpos[2] = gsign[newsym][2] * cpos[gperm[newsym][2]];

		nvecn[0] = gsign[newsym][0] * vecn[gperm[newsym][0]];
		nvecn[1] = gsign[newsym][1] * vecn[gperm[newsym][1]];
		nvecn[2] = gsign[newsym][2] * vecn[gperm[newsym][2]];

		

		if (wall[wh])
		{
			lcor[0] *= lfe[0];
			lcor[1] *= lfe[1];
			lcor[2] *= lfe[2];

			lcol = lcor[0] + lcor[1] + lcor[2];
			dark = (-1.0 / seedist[0]) * (coll[min]) + 1.0;
			if (dark < 0) dark = 0.0;
			lfac = (1.0 / (4 * lsize * lsize)) * lcol * lcol;
			if (lfac > 1.0) lfac = 1.0;

			if (lcor[0] > lcor[1]) lcormax = lcor[0];
			else lcormax = lcor[1];
			if (lcor[2] > lcormax) lcormax = lcor[2];

			lcorr = 255.0 * lcor[0] / lcormax;
			lcorg = 255.0 * lcor[1] / lcormax;
			lcorb = 255.0 * lcor[2] / lcormax;

			if (ncpos[0] == 0.5 || ncpos[0] == -0.5)
			{
				coordx = 100*(ncpos[1]+0.5);
				coordy = 100 * (ncpos[2]+0.5);
			}
			else if (ncpos[1] == 0.5 || ncpos[1] == -0.5)
			{
				coordx = 100 * (ncpos[0]+0.5);
				coordy = 100 * (ncpos[2]+0.5);
			}
			else
			{
				coordx = 100 * (ncpos[0]+0.5);
				coordy = 100 * (ncpos[1]+0.5);
			}

			coord = coordx + 100 * coordy;

			for (l = 0; l < 10; l++) coord = (60493 * coord + 11) % 479001599;
			coord %= 20;
			coord += 45;

			
				buffer[4 * tmp] = haz[0] * noise + (1 - haz[0]) * ((1 - lfac) * dark * 2.0*coord+ lfac * lcorr);
				buffer[4 * tmp + 1] = haz[0] * noise + (1 - haz[0]) * ((1 - lfac) * dark *coord + lfac * lcorg);
				buffer[4 * tmp + 2] = haz[0] * noise + (1 - haz[0]) * ( lfac * lcorb);
			
			

			return;
		}
	
		
	


		astcolmin = 2;
		qa = nvecn[0] * nvecn[0] + nvecn[1] * nvecn[1] + nvecn[2] * nvecn[2];
		for (i = 0; i < astnum[0] + astnumplus[0]; i++)
		{
			
			qb = 2 * (nvecn[0] * (ncpos[0] - astpos0[i]) + nvecn[1] * (ncpos[1] - astpos1[i]) + nvecn[2] * (ncpos[2] - astpos2[i]));
			qc = (ncpos[0] - astpos0[i]) * (ncpos[0] - astpos0[i]) + (ncpos[1] - astpos1[i]) * (ncpos[1] - astpos1[i]) + (ncpos[2] - astpos2[i]) * (ncpos[2] - astpos2[i]) - astrad[i] * astrad[i];
			
			discr = qb * qb - 4 * qa * qc;

	

			if (discr > 0)
			{
				t1 = ((-1.0) * qb - sqrt(discr)) / (2.0 * qa);
				if (t1 > 0 && t1<astcolmin)
				{
						astcolmin = t1;
						astcolidx = i;
				}
			}
		}

		for (i = 0; i < cnumplus[0]; i++)
		{
			qb = 2 * (nvecn[0] * (ncpos[0] - ccpos0[i]) + nvecn[1] * (ncpos[1] - ccpos1[i]) + nvecn[2] * (ncpos[2] - ccpos2[i]));
			qc = (ncpos[0] - ccpos0[i]) * (ncpos[0] - ccpos0[i]) + (ncpos[1] - ccpos1[i]) * (ncpos[1] - ccpos1[i]) + (ncpos[2] - ccpos2[i]) * (ncpos[2] - ccpos2[i]) - csize * csize;

			discr = qb * qb - 4 * qa * qc;


			if (discr > 0)
			{
				t1 = ((-1.0) * qb - sqrt(discr)) / (2.0 * qa);
				if (t1 > 0 && t1 < astcolmin)
				{
					astcolmin = t1;
					astcolidx = -1;
					ccolidx = i;
				}
			}
		}

		
		
		currentsym = newsym;

		lastcollmin = coll[min];
		coll[min] += inv[min];

		if (coll[0] < coll[1]) min = 0;
		else min = 1;
		if (coll[2] < coll[min]) min = 2;

		for (i = 0; i < 3; i++)
		{
			if (lout[i])
			{
				qb = 2 * (nvecn[0] * (ncpos[0] - lpos0[i]) + nvecn[1] * (ncpos[1] - lpos1[i]) + nvecn[2] * (ncpos[2] - lpos2[i]));
				qc = (ncpos[0] - lpos0[i]) * (ncpos[0] - lpos0[i]) + (ncpos[1] - lpos1[i]) * (ncpos[1] - lpos1[i]) + (ncpos[2] - lpos2[i]) * (ncpos[2] - lpos2[i]) - lsize * lsize;

				discr = qb * qb - 4 * qa * qc;


				if (discr > 0)
				{
					t1 = ((-1.0) * qb - sqrt(discr)) / (2.0 * qa);
					if (t1 > 0 && t1 < astcolmin && t1<coll[min]-lastcollmin)
					{
						lcor[i] += ((-1.0) * qb + sqrt(discr)) / (2.0 * qa) - t1;
					}
				}
			}
		}

		for (i = 0; i < lnumplus[0]; i++)
		{
			qb = 2 * (nvecn[0] * (ncpos[0] - lpos0[3 + i]) + nvecn[1] * (ncpos[1] - lpos1[3 + i]) + nvecn[2] * (ncpos[2] - lpos2[3 + i]));
			qc = (ncpos[0] - lpos0[3 + i]) * (ncpos[0] - lpos0[3 + i]) + (ncpos[1] - lpos1[3 + i]) * (ncpos[1] - lpos1[3 + i]) + (ncpos[2] - lpos2[3 + i]) * (ncpos[2] - lpos2[3 + i]) - lsize * lsize;

			discr = qb * qb - 4 * qa * qc;


			if (discr > 0)
			{
				t1 = ((-1.0) * qb - sqrt(discr)) / (2.0 * qa);
				if (t1 > 0 && t1 < astcolmin && t1 < coll[min] - lastcollmin)
				{
					lcor[lidx[3 + i]] += ((-1.0) * qb + sqrt(discr)) / (2.0 * qa) - t1;
				}
			}
		}

		if (astcolmin < coll[min] - lastcollmin)
		{
			lcor[0] *= lfe[0];
			lcor[1] *= lfe[1];
			lcor[2] *= lfe[2];

			lcol = lcor[0] + lcor[1] + lcor[2];
			dark = (-1.0 / seedist[0]) * (astcolmin + lastcollmin) + 1.0;
			if (dark < 0) dark = 0.0;
			lfac = (1.0 / (4 * lsize * lsize)) * lcol * lcol;
			if (lfac > 1.0) lfac = 1.0;

			if (lcor[0] > lcor[1]) lcormax = lcor[0];
			else lcormax = lcor[1];
			if (lcor[2] > lcormax) lcormax = lcor[2];

			lcorr = 255.0 * lcor[0] / lcormax;
			lcorg = 255.0 * lcor[1] / lcormax;
			lcorb = 255.0 * lcor[2] / lcormax;

			

			if (astcolidx == -1)
			{
					newsym = ccsym[ccolidx];

					cont[0] = (ncpos[0] + astcolmin * nvecn[0]) - ccpos0[ccolidx];
					cont[1] = (ncpos[1] + astcolmin * nvecn[1]) - ccpos1[ccolidx];
					cont[2] = (ncpos[2] + astcolmin * nvecn[2]) - ccpos2[ccolidx];

					conttmp[0] = gsign[newsym][0] * cont[gperm[newsym][0]];
					conttmp[1] = gsign[newsym][1] * cont[gperm[newsym][1]];
					conttmp[2] = gsign[newsym][2] * cont[gperm[newsym][2]];

					ncont0 = m0 * conttmp[0] + m1 * conttmp[1] + m2 * conttmp[2];
					ncont1 = m3 * conttmp[0] + m4 * conttmp[1] + m5 * conttmp[2];
					ncont2 = m6 * conttmp[0] + m7 * conttmp[1] + m8 * conttmp[2];

					ncont0 /= csize;
					ncont1 /= csize;
					ncont2 /= csize;

					u =(0.5 + atan2(ncont1, ncont0) / (2.0 * M_PI));
					v = (0.5 + asin(ncont2) / M_PI);
		
					rnbwv = rnbw2(u);
					colr = rnbwv % 256;
					rnbwv -= colr;
					rnbwv /= 256;
					colg = rnbwv % 256;
					rnbwv -= colg;
					rnbwv /= 256;
					col = rnbwv % 256;
					
					if (v > 0.5)
					{
						vf = 2.0 * v - 1.0;

						ccor0 = (1-vf)*colr + vf*255.0;
						ccor1 = (1 - vf) * colg + vf * 255.0;
						ccor2 = (1 - vf) * col + vf * 255.0;
					}
					else
					{
						vf = 1.0 - 2.0 * v;

						ccor0 = (1 - vf) * colr;
						ccor1 = (1 - vf) * colg;
						ccor2 = (1 - vf) * col;
					}

					if ((u-0.5)*(u-0.5) + (v-0.5)*(v-0.5)<0.0035)
					{
						ccor0 = 128;
						ccor1 = 128;
						ccor2 = 128;
					}
					if ((u - 0.5) * (u - 0.5) + (v - 0.5) * (v - 0.5) < 0.0025)
					{
						ccor0 = 0;
						ccor1 = 0;
						ccor2 = 0;
					}


					buffer[4 * tmp] = haz[0] * noise + (1 - haz[0]) * ((1 - lfac) * dark * (ccor0) + lfac * lcorr);
					buffer[4 * tmp + 1] = haz[0] * noise + (1 - haz[0]) * ((1 - lfac) * dark * (ccor1)+lfac * lcorg);
					buffer[4 * tmp + 2] = haz[0] * noise + (1 - haz[0]) * ((1 - lfac) * dark * (ccor2)+lfac * lcorb);

				
			}
			else {
				newsym = astsym[astcolidx];
				newsym2 = topo[currenttopo][asttopo[astplusidx[astcolidx]]];

				cont[0] = (ncpos[0] + astcolmin * nvecn[0]) - astpos0[astcolidx];
				cont[1] = (ncpos[1] + astcolmin * nvecn[1]) - astpos1[astcolidx];
				cont[2] = (ncpos[2] + astcolmin * nvecn[2]) - astpos2[astcolidx];

				conttmp[0] = gsign[newsym][0] * cont[gperm[newsym][0]];
				conttmp[1] = gsign[newsym][1] * cont[gperm[newsym][1]];
				conttmp[2] = gsign[newsym][2] * cont[gperm[newsym][2]];

				cont[0] = gsign[newsym2][0] * conttmp[gperm[newsym2][0]];
				cont[1] = gsign[newsym2][1] * conttmp[gperm[newsym2][1]];
				cont[2] = gsign[newsym2][2] * conttmp[gperm[newsym2][2]];


				ncont0 = astmat[astplusidx[astcolidx]][0] * cont[0] + astmat[astplusidx[astcolidx]][1] * cont[1] + astmat[astplusidx[astcolidx]][2] * cont[2];
				ncont1 = astmat[astplusidx[astcolidx]][3] * cont[0] + astmat[astplusidx[astcolidx]][4] * cont[1] + astmat[astplusidx[astcolidx]][5] * cont[2];
				ncont2 = astmat[astplusidx[astcolidx]][6] * cont[0] + astmat[astplusidx[astcolidx]][7] * cont[1] + astmat[astplusidx[astcolidx]][8] * cont[2];

				ncont0 /= astrad[astplusidx[astcolidx]];
				ncont1 /= astrad[astplusidx[astcolidx]];
				ncont2 /= astrad[astplusidx[astcolidx]];

				u = 100.0 * (0.5 + atan2(ncont1, ncont0) / (2.0 * M_PI));
				v = 100.0 * (0.5 + asin(ncont2) / M_PI);

				uv = (int)u + 100 * (int)v;

				for (l = 0; l < 10; l++) uv = (60493 * uv + 11) % 479001599;

				uv %= 30;
				uv += 128;

				buffer[4 * tmp] = haz[0] * noise + (1 - haz[0]) * ((1 - lfac) * dark * uv + lfac * lcorr);
				buffer[4 * tmp + 1] = haz[0] * noise + (1 - haz[0]) * ((1 - lfac) * dark * uv + lfac * lcorg);
				buffer[4 * tmp + 2] = haz[0] * noise + (1 - haz[0]) * ((1 - lfac) * dark * uv + lfac * lcorb);
			}

			return;
		}
	}

	lcor[0] *= lfe[0];
	lcor[1] *= lfe[1];
	lcor[2] *= lfe[2];

	lcol = lcor[0] + lcor[1] + lcor[2];
	lfac = (1.0 / (4 * lsize * lsize)) * lcol * lcol;
	if (lfac > 1.0) lfac = 1.0;

	if (lcor[0] > lcor[1]) lcormax = lcor[0];
	else lcormax = lcor[1];
	if (lcor[2] > lcormax) lcormax = lcor[2];

	lcorr = 255.0 * lcor[0] / lcormax;
	lcorg = 255.0 * lcor[1] / lcormax;
	lcorb = 255.0 * lcor[2] / lcormax;

	buffer[4 * tmp] = haz[0] * noise + (1 - haz[0]) * (lcorr * lfac);
	buffer[4 * tmp + 1] = haz[0] * noise + (1 - haz[0]) * (lcorg*lfac);
	buffer[4 * tmp + 2] = haz[0] * noise + (1 - haz[0]) * (lcorb * lfac);


}


void cudaInit(int syms[48][6], int gstruct[48][48], int perm[48][3], int sign[48][3], int gtopo[187][1728], int ginv[48])
{
	
	int i;
	double disto = 2;
	double sqsz = 0.01 / 6;
	int tmpx, tmpy;

	double* vecltmp = new double[1920 * 1080];

	double vec0, vec1, vec2;
	double addy0, addy1, addy2;
	double addz0, addz1, addz2;
	double vecn0, vecn1, vecn2;
	double x00 = 1, x01 = 0, x02 = 0;
	double x10 = 0, x11 = 1, x12 = 0;
	double x20 = 0, x21 = 0, x22 = 1;
	double multy = (1 - 1920) * sqsz / 2;
	double multz = (1080 - 1) * sqsz / 2;

	int* hsyms[187]{};
	double* hhsyms[64]{};

	hipSetDevice(0);
	hipMalloc((void**)&buffer, 4 * 1920 * 1080 * sizeof(uint8_t));
	hipMalloc((void**)&vecl, 1920 * 1080 * sizeof(double));
	
	hipMalloc((void**)&gsyms,  48* sizeof(int*));
	for (i = 0; i < 48; i++) hipMalloc((void**)&hsyms[i], 6 * sizeof(int));
	hipMemcpy(gsyms, hsyms, 48 * sizeof(int*), hipMemcpyHostToDevice);
	for (i = 0; i < 48; i++)  hipMemcpy(hsyms[i], syms[i], 6 * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&ggstruct, 48 * sizeof(int*));
	for (i = 0; i < 48; i++) hipMalloc((void**)&hsyms[i], 48 * sizeof(int));
	hipMemcpy(ggstruct, hsyms, 48 * sizeof(int*), hipMemcpyHostToDevice);
	for (i = 0; i < 48; i++)  hipMemcpy(hsyms[i], gstruct[i], 48 * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&gperm, 48 * sizeof(int*));
	for (i = 0; i < 48; i++) hipMalloc((void**)&hsyms[i], 3 * sizeof(int));
	hipMemcpy(gperm, hsyms, 48 * sizeof(int*), hipMemcpyHostToDevice);
	for (i = 0; i < 48; i++)  hipMemcpy(hsyms[i], perm[i], 3 * sizeof(int), hipMemcpyHostToDevice);
	
	hipMalloc((void**)&gsign, 48 * sizeof(int*));
	for (i = 0; i < 48; i++) hipMalloc((void**)&hsyms[i], 3 * sizeof(int));
	hipMemcpy(gsign, hsyms, 48 * sizeof(int*), hipMemcpyHostToDevice);
	for (i = 0; i < 48; i++)  hipMemcpy(hsyms[i], sign[i], 3 * sizeof(int), hipMemcpyHostToDevice);
	
	hipMalloc((void**)&topo, 187 * sizeof(int*));
	for (i = 0; i < 187; i++) hipMalloc((void**)&hsyms[i], 1728 * sizeof(int));
	hipMemcpy(topo, hsyms, 187 * sizeof(int*), hipMemcpyHostToDevice);
	for (i = 0; i < 187; i++)  hipMemcpy(hsyms[i], gtopo[i], 1728 * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&astmat, 64 * sizeof(double*));
	for (i = 0; i < 64; i++) hipMalloc((void**)&hhsyms[i], 9 * sizeof(double));
	hipMemcpy(astmat, hhsyms, 64 * sizeof(double*), hipMemcpyHostToDevice);

	hipMalloc((void**)&astmatv, 64 * sizeof(double*));
	for (i = 0; i < 64; i++) hipMalloc((void**)&hhsyms[i], 9 * sizeof(double));
	hipMemcpy(astmatv, hhsyms, 64 * sizeof(double*), hipMemcpyHostToDevice);
	

	vec0 = disto * x00 + multy * x10 + multz * x20;
	vec1 = disto * x01 + multy * x11 + multz * x21;
	vec2 = disto * x02 + multy * x12 + multz * x22;

	addy0 = sqsz * x10;
	addy1 = sqsz * x11;
	addy2 = sqsz * x12;

	addz0 = -sqsz * x20;
	addz1 = -sqsz * x21;
	addz2 = -sqsz * x22;

	for (i = 0; i < 1920 * 1080; i++)
	{
		tmpx = i % 1920;
		tmpy = (i - tmpx) / 1920;

		vecn0 = vec0 + tmpx * addy0 + tmpy * addz0;
		vecn1 = vec1 + tmpx * addy1 + tmpy * addz1;
		vecn2 = vec2 + tmpx * addy2 + tmpy * addz2;

		vecltmp[i] = 1.0/sqrt(vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2);
	}

	hipMemcpy(vecl, vecltmp, 1920 * 1080 * sizeof(double), hipMemcpyHostToDevice);

	

	hipMalloc((void**)&astrad, 64 * sizeof(double));
	hipMalloc((void**)&asttopo, 64 * sizeof(int));
	hipMalloc((void**)&astpos0, 64 * sizeof(double));
	hipMalloc((void**)&astpos1, 64 * sizeof(double));
	hipMalloc((void**)&astpos2, 64 * sizeof(double));
	hipMalloc((void**)&astv0, 64 * sizeof(double));
	hipMalloc((void**)&astv1, 64 * sizeof(double));
	hipMalloc((void**)&astv2, 64 * sizeof(double));
	hipMalloc((void**)&astnumplus, sizeof(int));
	hipMalloc((void**)&cnumplus, sizeof(int));
	hipMalloc((void**)&ccpos0, 10 * sizeof(double));
	hipMalloc((void**)&ccpos1, 10 * sizeof(double));
	hipMalloc((void**)&ccpos2, 10 * sizeof(double));
	hipMalloc((void**)&ccsym, 10 * sizeof(int));
	hipMalloc((void**)&astsym, 64 * sizeof(int));

	hipMalloc((void**)&lnumplus, sizeof(int));
	hipMalloc((void**)&lpos0, 64 * sizeof(double));
	hipMalloc((void**)&lpos1, 64 * sizeof(double));
	hipMalloc((void**)&lpos2, 64 * sizeof(double));
	hipMalloc((void**)&lv0, 64 * sizeof(double));
	hipMalloc((void**)&lv1, 64 * sizeof(double));
	hipMalloc((void**)&lv2, 64 * sizeof(double));
	hipMalloc((void**)&lout, 64 * sizeof(bool));
	hipMalloc((void**)&loutf, 64 * sizeof(int));
	hipMalloc((void**)&lidx, 64 * sizeof(int));
	hipMalloc((void**)&astnum, sizeof(int));
	hipMalloc((void**)&astplusidx, 64 * sizeof(int));
	hipMalloc((void**)&rip, sizeof(bool));
	hipMalloc((void**)&haz, sizeof(double));
	hipMalloc((void**)&seedist, sizeof(double));
	hipMalloc((void**)&dist, sizeof(double));
	hipMalloc((void**)&gginv, 48*sizeof(int));
	hipMemcpy(gginv, ginv, 48 * sizeof(int), hipMemcpyHostToDevice);

	setast << <1, 1 >> > (buffer, astrad, astpos0, astpos1, astpos2,astv0,astv1,astv2,astnum,seedist,dist,astmat,astmatv);
	hipDeviceSynchronize();
}

void cudaExit()
{
	hipFree(buffer);
	hipFree(vecl);
	hipFree(astrad);
	hipFree(astpos0);
	hipFree(astpos1);
	hipFree(astpos2);
	hipFree(astv0);
	hipFree(astv1);
	hipFree(astv2);
	hipFree(astnumplus);
	hipFree(cnumplus);
	hipFree(ccpos0);
	hipFree(ccpos1);
	hipFree(ccpos2);
	hipFree(lnumplus);
	hipFree(lpos0);
	hipFree(lpos1);
	hipFree(lpos2);
	hipFree(lv0);
	hipFree(lv1);
	hipFree(lv2);
	hipFree(lout);
	hipFree(loutf);
	hipFree(astnum);
	hipFree(astplusidx);
	hipFree(gginv);
	hipFree(seedist);
	hipFree(dist);
	hipFree(ccsym);
	hipFree(astsym);
	hipFree(astmat);
	hipFree(astmatv);
	hipDeviceReset();
}

void cudathingy(uint8_t* pixels, double pos0, double pos1, double pos2, double vec0, double vec1, double vec2, double addy0, double addy1, double addy2, double addz0, double addz1, double addz2, int rep0, int rep1, int rep2, int rep3, int rep4, int rep5, int currenttopo, bool fireflag, double v0, double v1, double v2, int nbframe, bool resetflag, bool sdp, bool sdm, bool fovp, bool fovm, double cv0, double cv1, double cv2, bool w0, bool w1, bool w2, bool w3, bool w4, bool w5, double m0, double m1, double m2, double m3, double m4, double m5, double m6, double m7, double m8)
{
	if (fovp || fovm || resetflag)
	{
		modfovp << <1, 1>>> (dist, fovp, fovm, resetflag);
		hipDeviceSynchronize();
		modfov << <(int)(1920 * 1080 / 480), 480 >> > (dist, vecl);
		hipDeviceSynchronize();
	}
	rotast << <1, 32 >> > (astmat, astmatv);
	hipDeviceSynchronize();
	modast << <1, 1 >> > (astrad, astpos0, astpos1, astpos2,astv0,astv1,astv2,topo,gperm,gsign,currenttopo,astnum,astnumplus, lpos0,lpos1,lpos2,lv0,lv1,lv2,lnumplus, pos0, pos1, pos2, ccpos0, ccpos1, ccpos2, cnumplus, fireflag, v0, v1, v2, lout, loutf, nbframe,gginv,astplusidx,rip,haz,lidx,resetflag,seedist,sdp,sdm,cv0,cv1,cv2,w0,w1,w2,w3,w4,w5,ccsym,astsym,astmat,astmatv,asttopo);
	hipDeviceSynchronize();
	addKernel << <(int)(1920 * 1080 / 480), 480 >> > (buffer, gsyms, ggstruct, gperm, gsign, vecl, pos0, pos1, pos2, vec0, vec1, vec2, addy0, addy1, addy2, addz0, addz1, addz2,rep0,rep1,rep2,rep3,rep4,rep5, astnum, astnumplus, astrad,  astpos0,  astpos1,  astpos2,currenttopo,ccpos0, ccpos1,  ccpos2, cnumplus, lpos0, lpos1, lpos2, lnumplus,rip,nbframe,haz,lidx,lout,seedist,w0,w1,w2,w3,w4,w5,loutf,m0,m1,m2,m3,m4,m5,m6,m7,m8,ccsym,astsym,astmat,astplusidx,asttopo,topo);
	hipDeviceSynchronize();
	hipMemcpy(pixels, buffer, 4 * 1920 * 1080 * sizeof(uint8_t), hipMemcpyDeviceToHost);
}